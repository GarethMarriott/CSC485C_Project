#include "hip/hip_runtime.h"



#include <iostream>
#include <stdio.h>
#include <fstream>
#include <sstream>
#include <vector>
#include <queue>
#include <omp.h>
#include <chrono>
#include <numeric>
// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>

using namespace std;

int const blocksize = 512;




class graph
{
   int n;
   // thrust::host_vector< thrust::host_vector<int> > distance;
   // thrust::host_vector< thrust::host_vector<int> > path;
   // thrust::host_vector< thrust::host_vector<int> > adjacency_list;
   // thrust::host_vector<int> adjacency_offset;
   // thrust::host_vector<int> adjacency_size;
   // thrust::host_vector< thrust::host_vector<bool> > discovered;
   vector< vector<int> > distance;
   vector< vector<int> > path;
   vector< vector<int> > adjacency_list;
   vector<int> adjacency_offset;
   vector<int> adjacency_size;
   vector< vector<bool> > discovered;
   public:
        void get_data(std::string filename);
        void bfs();
        void print();
        void print_path(int start, int goal);
};

// struct node
// {
//     int value;
//     int parent;
//     int depth;
//     node(int v, int p, int d)
//     {
//         value = v;
//         parent = p;
//         depth = d;
//     }
//     node(){
//       value = 0;
//       parent = 0;
//       depth = 0;
//     }
// };
__device__
struct node
{
    int value;
    int parent;
    int depth;
    // node(int v, int p, int d)
    // {
    //     value = v;
    //     parent = p;
    //     depth = d;
    // }
    // node(){
    //   value = 0;
    //   parent = 0;
    //   depth = 0;
    // }
};

__global__
void process_row( int *dev_adjacency_list , bool *dev_discovered , int *dev_path , int *dev_distance , int *dev_adjacency_offset , int *dev_adjacency_size , size_t n )
{



  int const idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx<n) {

    // if(idx<1){
    //   for(int i=0; i<n; i++){
    //     for(int j=0; j<n; j++){
    //       printf("%d ", dev_path[i*n+j]);
    //     }
    //     printf("\n");
    //   }
    // }

    struct node* queue = (struct node*)malloc(sizeof(struct node)*n);
    int head;
    int tail;
    struct node curr;

    head = 0;
    tail = 0;

    for(int j=0; j<dev_adjacency_size[idx]; j++)
    {
        queue[tail++] = node{dev_adjacency_list[dev_adjacency_offset[idx]+j], idx, 1};
        dev_discovered[idx*n + dev_adjacency_list[dev_adjacency_offset[idx]+j]] = true;
    }


    while(head != tail)
    {
        curr = queue[head++];

        dev_path[idx*n + curr.value] = curr.parent;
        dev_distance[idx*n + curr.value] = curr.depth;

        for(int j=0; j<dev_adjacency_size[curr.value]; j++)
        {
            if(!dev_discovered[idx*n + dev_adjacency_list[dev_adjacency_offset[curr.value]+j]])
            {
                queue[tail++] = node{dev_adjacency_list[dev_adjacency_offset[curr.value]+j], curr.value, curr.depth + 1};
                dev_discovered[idx*n + dev_adjacency_list[dev_adjacency_offset[curr.value]+j]] = true;
            }
        }
      }
    free(queue);
    }
}

void graph::get_data(std::string filename)
{
  /*string line;
  ifstream file ("test_matrix.txt");
  while(getline(file,line)){
      for(i=1;i<5;i++)
        adjacency[i][j] = (int)line;
  }


  printf("\nEnter the total number of vertices: ");
  scanf("%d",&n);
  printf("\nEnter the adjacency matrix (If there is no \nedge betweent two vertex then enter 1000): \n\n");
  for(i=1;i<=n;i++)
  {
       for(j=1;j<=n;j++)
       {
             scanf("%d", &adjacency[i][j]);
       }
  }*/

    string curr_row;

    // std::cout << typeid(adjacency_list).name() << '\n';

    ifstream f(filename);

    std::getline(f, curr_row);
    n = stoi(curr_row);

    // for (int i = 0; i < n; i++) {
    //   for (int j = 0; j < adjacency_list[i].size(); j++) {
		//       std::cout << adjacency_list[i][j] << " ";
	  //   }
    //   std::cout << " " << '\n';
	  // }

    adjacency_list.resize(n);
    distance.resize(n);
    path.resize(n);
    discovered.resize(n);

    adjacency_offset.resize(n);
    adjacency_size.resize(n);

    int adj_size = 0;

    for(int i=0; getline(f, curr_row); i++){
        // if (i%10 == 0) {
        //   printf("%d\n", i);
        // }

        adjacency_offset[i] = adj_size;

        std::stringstream ss(curr_row);
        while(getline(ss, curr_row, ' ')){
            adjacency_list[i].push_back(stoi(curr_row));
            adj_size++;
            //adjacency[i].push_back(stoi(curr_row));
        }

        adjacency_size[i] = adj_size - adjacency_offset[i];

    }

    //#pragma omp parallel for
    for(int i=0; i<n; i++){
        path[i].resize(n, -1);
        distance[i].resize(n, -1);
        discovered[i].resize(n, false);
    }

    // #pragma omp parallel for
    // for(int i=0; i<n; i++){
    //     for(int j=0; j<n; j++){
    //         path[i][j] = -1;
    //         distance[i][j] = -1;
    //         discovered[i][j] = false;
    //     }
    // }

    //#pragma omp parallel for
    for(int i=0; i<n; i++){
        path[i][i] = i;
        distance[i][i] = 0;
        discovered[i][i] = true;
    }

}


void graph::print_path(int start, int finish)
{

    if(path[start][finish] == -1)
    {
        printf("\n\nThere is no path from %d to %d\n", start, finish);
        return;
    }

    vector< int > curr_path;

    int curr = finish;

    while(curr != start)
    {
        curr_path.push_back(curr);
        curr = path[start][curr];
    }

    curr_path.push_back(curr);

    printf("\n\nShortest path from %d to %d is:\n", start, finish);
    for(int i=curr_path.size() - 1; i>0; i--){
        printf("%d, ", curr_path[i]);
    }
    printf("%d\n", curr_path[0]);
}


void graph::bfs()
{
    auto const size = sizeof(int) * n;
    auto const num_blocks = ceil( n / static_cast< float >( blocksize ) );

    int E = std::accumulate(adjacency_size.begin(), adjacency_size.end(), 0);

    int* adjacency_list_array = (int*)malloc(sizeof(int)*E);
    int index = 0;
    for (int i = 0; i < adjacency_list.size(); i++) {
      for (int j = 0; j < adjacency_list[i].size(); j++) {
        adjacency_list_array[index] = adjacency_list[i][j];
        index++;
      }
    }

    bool* discovered_array = (bool*)malloc(sizeof(bool)*n*n);
    int* path_array        = (int*)malloc(sizeof(int)*n*n);
    int* distance_array    = (int*)malloc(sizeof(int)*n*n);
    for (int i = 0; i < n; i++) {
      for (int j = 0; j < n; j++) {
        discovered_array[i*n+j] = discovered[i][j];
        path_array[i*n+j]       = path[i][j];
        distance_array[i*n+j]   = distance[i][j];
      }
    }

    int* adjacency_offset_array = (int*)malloc(sizeof(int)*n);
    int* adjacency_size_array   = (int*)malloc(sizeof(int)*n);

    for (int i = 0; i < n; i++) {
      adjacency_offset_array[i] = adjacency_offset[i];
      adjacency_size_array[i]   = adjacency_size[i];
    }

    // for(int i=0; i<n; i++){
    //   for(int j=0; j<n; j++){
    //     printf("%d ", path_array[i*n+j]);
    //   }
    //   printf("\n");
    // }

    int* dev_adjacency_list;
    bool* dev_discovered;
    int* dev_path;
    int* dev_distance;
    int* dev_adjacency_offset;
    int* dev_adjacency_size;

    hipMalloc( (void **) &dev_adjacency_list, sizeof(int)*E);
    hipMalloc( (void **) &dev_discovered, sizeof(bool)*n*n);
    hipMalloc( (void **) &dev_path, sizeof(int)*n*n);
    hipMalloc( (void **) &dev_distance, sizeof(int)*n*n);
    hipMalloc( (void **) &dev_adjacency_offset, sizeof(int)*n);
    hipMalloc( (void **) &dev_adjacency_size, sizeof(int)*n);

    hipMemcpy( dev_adjacency_list, adjacency_list_array, sizeof(int)*E, hipMemcpyHostToDevice );
    hipMemcpy( dev_discovered, discovered_array, sizeof(bool)*n*n, hipMemcpyHostToDevice );
    hipMemcpy( dev_path, path_array, sizeof(int)*n*n, hipMemcpyHostToDevice );
    hipMemcpy( dev_distance, distance_array, sizeof(int)*n*n, hipMemcpyHostToDevice );
    hipMemcpy( dev_adjacency_offset, adjacency_offset_array, sizeof(int)*n, hipMemcpyHostToDevice );
    hipMemcpy( dev_adjacency_size, adjacency_size_array, sizeof(int)*n, hipMemcpyHostToDevice );

    process_row<<< num_blocks, blocksize >>>( dev_adjacency_list, dev_discovered, dev_path, dev_distance, dev_adjacency_offset, dev_adjacency_size, n );

    hipMemcpy( path_array, dev_path, sizeof(int)*n*n, hipMemcpyDeviceToHost );
    hipMemcpy( distance_array, dev_distance, sizeof(int)*n*n, hipMemcpyDeviceToHost );

    for(int i=0; i<n; i++){
      for(int j=0; j<n; j++){
        path[i][j] = path_array[i*n+j];
        distance[i][j] = distance_array[i*n+j];
      }
    }
    // thrust::device_vector< thrust::device_vector< int > > d_adjacency_list = adjacency_list;
    // thrust::device_vector< thrust::device_vector< bool > > d_discovered = discovered;
    // thrust::device_vector< thrust::device_vector< int > > d_path = path;
    // thrust::device_vector< thrust::device_vector< int > > d_distance = distance;
    // thrust::device_vector< int > d_adjacency_offset = adjacency_offset;
    // thrust::device_vector< int > d_adjacency_size = adjacency_size;
    //
    // int *dev_adjacency_list = thrust::raw_pointer_cast(&d_adjacency_list[0][0]);
    // bool *dev_discovered = thrust::raw_pointer_cast(d_discovered[0].data());
    // int *dev_path = thrust::raw_pointer_cast(d_path[0].data());
    // int *dev_distance = thrust::raw_pointer_cast(d_distance[0].data());
    // int *dev_adjacency_offset = thrust::raw_pointer_cast(d_adjacency_offset.data());
    // int *dev_adjacency_size = thrust::raw_pointer_cast(d_adjacency_size.data());


    //adjacency_list = dev_adjacency_list;
    //discovered = dev_discovered;

}



void graph::print()
{

  printf("\n\nThe adjacency list is: \n\n");
  for(int i=0;i<n;i++)
  {
      for(uint j=0;j<adjacency_list[i].size();j++)
      {
          printf("%d\t",adjacency_list[i][j]);
      }
      printf("\n");
  }

  printf("\n\nThe distance matrix is: \n\n");
  for(int i=0;i<n;i++)
  {
      for(int j=0;j<n;j++)
      {
          printf("%d\t",distance[i][j]);
      }
      printf("\n");
  }

  printf("\n\nThe final path matrix is: \n\n");
  for(int i=0;i<n;i++)
  {
      for(int j=0;j<n;j++)
      {
          printf("%d\t",path[i][j]);
      }
      printf("\n");
  }
}

int main(int argc, char const *argv[])
{
  std::string filename(argv[1]);
  int num_threads = stoi(argv[2]);
  // omp_set_num_threads(num_threads);

  auto full_start = std::chrono::steady_clock::now();

  graph graph;
  graph.get_data(filename);
  auto start = std::chrono::steady_clock::now();
  graph.bfs();
  auto end = std::chrono::steady_clock::now();
  std::chrono::duration<long double> time = end - start;
  std::cout << "BFS run time : " << time.count() << std::endl;

  std::chrono::duration<long double> full_time = end - full_start;
  std::cout << "BFS + preprocessing run time : " << full_time.count() << std::endl;

  graph.print();
  // graph.print_path(2, 1);
  return 0;
}
