#include "hip/hip_runtime.h"



#include <iostream>
#include <stdio.h>
#include <fstream>
#include <sstream>
#include <vector>
#include <queue>
#include <omp.h>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

int const blocksize = 512;




class graph
{
   int n;
   thrust::host_vector< thrust::host_vector<int> > distance;
   thrust::host_vector< thrust::host_vector<int> > path;
   thrust::host_vector< thrust::host_vector<int> > adjacency_list;
   thrust::host_vector< thrust::host_vector<bool> > discovered;
   public:
        void get_data(std::string filename);
        void bfs();
        void print();
        void print_path(int start, int goal);
};

struct node
{
    int value;
    int parent;
    int depth;
    node(int v, int p, int d)
    {
        value = v;
        parent = p;
        depth = d;
    }
};

__global__
void process_row( float *dev_adjacency_list , float *dev_discovered , float *dev_path , float *dev_distance , size_t n )
{
  int const i = threadIdx.x + blockIdx.x * blockDim.x;

  const struct node queue[n];
  int head;
  int tail;
  struct node curr;
  for(int i=0; i<n; i++)
  {
      //if (i%10 == 0) {
      //  cout << "\r" << i << std::flush;
      //}

      head = 0;
      tail = 0;

      for(int j=0; j<adjacency_list[i].size(); j++)
      {
          queue[tail++] = node(adjacency_list[i][j], i, 1);
          discovered[i][adjacency_list[i][j]] = true;
      }


      while(head != tail)
      {
          curr = queue[head++];

          path[i][curr.value] = curr.parent;
          distance[i][curr.value] = curr.depth;

          for(int j=0; j<adjacency_list[curr.value].size(); j++)
          {
              if(!discovered[i][adjacency_list[curr.value][j]])
              {
                  queue[tail++] = node(adjacency_list[curr.value][j], curr.value, curr.depth + 1);
                  discovered[i][adjacency_list[curr.value][j]] = true;
              }
          }
      }
  }

}

void graph::get_data(std::string filename)
{
  /*string line;
  ifstream file ("test_matrix.txt");
  while(getline(file,line)){
      for(i=1;i<5;i++)
        adjacency[i][j] = (int)line;
  }


  printf("\nEnter the total number of vertices: ");
  scanf("%d",&n);
  printf("\nEnter the adjacency matrix (If there is no \nedge betweent two vertex then enter 1000): \n\n");
  for(i=1;i<=n;i++)
  {
       for(j=1;j<=n;j++)
       {
             scanf("%d", &adjacency[i][j]);
       }
  }*/

    string curr_row;

    // std::cout << typeid(adjacency_list).name() << '\n';

    ifstream f(filename);

    std::getline(f, curr_row);
    n = stoi(curr_row);

    // for (int i = 0; i < n; i++) {
    //   for (int j = 0; j < adjacency_list[i].size(); j++) {
		//       std::cout << adjacency_list[i][j] << " ";
	  //   }
    //   std::cout << " " << '\n';
	  // }

    adjacency_list.resize(n);
    distance.resize(n);
    path.resize(n);
    discovered.resize(n);

    for(int i=0; getline(f, curr_row); i++){
        // if (i%10 == 0) {
        //   printf("%d\n", i);
        // }
        std::stringstream ss(curr_row);
        int j=0;
        while(getline(ss, curr_row, ' ')){
            if(stoi(curr_row) > 0){
                adjacency_list[i].push_back(j);
            }
            //adjacency[i].push_back(stoi(curr_row));
            path[i].push_back(-1);
            distance[i].push_back(-1);
            discovered[i].push_back(false);
            j++;
        }
    }

    for(int i=0; i<n; i++){
        path[i][i] = i;
        distance[i][i] = 0;
        discovered[i][i] = true;
    }

}


void graph::print_path(int start, int finish)
{

    if(path[start][finish] == -1)
    {
        printf("\n\nThere is no path from %d to %d\n", start, finish);
        return;
    }

    vector< int > curr_path;

    int curr = finish;

    while(curr != start)
    {
        curr_path.push_back(curr);
        curr = path[start][curr];
    }

    curr_path.push_back(curr);

    printf("\n\nShortest path from %d to %d is:\n", start, finish);
    for(int i=curr_path.size() - 1; i>0; i--){
        printf("%d, ", curr_path[i]);
    }
    printf("%d\n", curr_path[0]);
}


void graph::bfs()
{
    auto const size = sizeof(int) * n;


    auto const num_blocks = ceil( n / static_cast< float >( blocksize ) );
    float *dev_row, *dev_result;

    float result[ n ];

    thrust::device_vector< thrust::device_vector< int > > dev_adjacency_list = adjacency_list;
    thrust::device_vector< thrust::device_vector< int > > dev_discovered = discovered;
    thrust::device_vector< thrust::device_vector< int > > dev_path = path;
    thrust::device_vector< thrust::device_vector< int > > dev_distance = distance;

    process_row<<< num_blocks, blocksize >>>( dev_adjacency_list, dev_discovered, dev_path, dev_distance, n );

    adjacency_list = dev_adjacency_list;
    discovered = dev_discovered;
    distance = dev_distance;
    path = dev_path;
}



void graph::print()
{

  printf("\n\nThe adjacency list is: \n\n");
  for(int i=0;i<n;i++)
  {
      for(uint j=0;j<adjacency_list[i].size();j++)
      {
          printf("%d\t",adjacency_list[i][j]);
      }
      printf("\n");
  }

  printf("\n\nThe distance matrix is: \n\n");
  for(int i=0;i<n;i++)
  {
      for(int j=0;j<n;j++)
      {
          printf("%d\t",distance[i][j]);
      }
      printf("\n");
  }

  printf("\n\nThe final path matrix is: \n\n");
  for(int i=0;i<n;i++)
  {
      for(int j=0;j<n;j++)
      {
          printf("%d\t",path[i][j]);
      }
      printf("\n");
  }
}

int main(int argc, char const *argv[])
{
  std::string filename(argv[1]);
  int num_threads = stoi(argv[2]);
  // omp_set_num_threads(num_threads);

  auto full_start = std::chrono::steady_clock::now();

  graph graph;
  graph.get_data(filename);
  auto start = std::chrono::steady_clock::now();
  graph.bfs();
  auto end = std::chrono::steady_clock::now();
  std::chrono::duration<long double> time = end - start;
  std::cout << "BFS run time : " << time.count() << std::endl;

  std::chrono::duration<long double> full_time = end - full_start;
  std::cout << "BFS + preprocessing run time : " << full_time.count() << std::endl;

  // graph.print();
  // graph.print_path(2, 1);
  return 0;
}
